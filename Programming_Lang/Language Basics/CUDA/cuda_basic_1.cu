
#include <hip/hip_runtime.h>
#include "stdio.h"
__global__ void add(int a, int b, int *c)
{
*c = a + b;
}
int main()
{
	int a,b,c;
	int *dev_c;
	a=3;
	b=4;
	hipMalloc((void**)&dev_c, sizeof(int));	//create malloc space in device; passed pointer (address of pointer in sent); address in gpu created will be pointed address
	//		dev_c        -|->               created_var
	//		121					155
	//	we are passing addr of pointer 121 but device creates memory with 155
	add<<<1,1>>>(a,b,dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d + %d is %d\n", a, b, c);
	hipFree(dev_c);
	
	//ERROR DETECTION CODE
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess)
	{
	   fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
	   exit(-1);
	}
	return 0;
}
