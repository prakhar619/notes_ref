
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

int main()
{
    hipDeviceProp_t prop;
    int dev;

    //cudaGetDevice(&dev);

    //memset(&prop,0,sizeof(cudaDeviceProp));
    prop.major = 1;
    prop.minor = 3;

    hipChooseDevice(&dev,&prop);

    printf("ID of CUDA device closed to revision 1.3:%d\n",dev);

    hipSetDevice(dev);
}
