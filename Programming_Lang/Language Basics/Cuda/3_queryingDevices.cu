#include <iostream>
using namespace std;

int main()
{
    int count;
    hipGetDeviceCount(&count);
    printf("No of Devices:%d\n",count);
    printf("\n");
    hipDeviceProp_t prop;

    for(int i=0; i < count; i++)
    {
        hipGetDeviceProperties(&prop,i);

        printf("General Info for device %d\n",i);
        printf("Name:%s\n",prop.name);
        printf("Compute Capabilities:%d.%d\n",prop.major,prop.minor);
        printf("Clock Rate:%d\n",prop.clockRate);
        printf("Total Global Memory:%ld\n",prop.totalGlobalMem);
        printf("Total Constant Memory:%ld\n",prop.totalConstMem);
        printf("Texture Alignment:%ld\n",prop.textureAlignment);
        printf("Multiprocessor count:%d\n",prop.multiProcessorCount);
        printf("Shared Mem per mp:%ld\n",prop.sharedMemPerBlock);
        printf("Registers per mp:%ld\n",prop.regsPerBlock);
        printf("Threads in warp:%d\n",prop.warpSize);
        printf("Max Threads per block:%d\n",prop.maxThreadsPerBlock);
        printf("Max Thread dimensions:(%d,%d,%d)\n",prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
        printf("Max Grid Dimensions:(%d,%d,%d)\n",prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
        printf("\n");
    }
    return 1;
}
