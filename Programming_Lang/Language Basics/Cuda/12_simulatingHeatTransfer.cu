#include "hip/hip_runtime.h"
//missing continue for here//
#include <iostream>

using namespace std;

#define DIM 1024
#define PI 3.1415
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.00001f
#define SPEED 0.25f

struct DataBlock
{
    unsigned char* output_bitmap;
    float *dev_inSrc;
    float *dev_outSrc;
    float *dev_constSrc;
    CPUAnimBitmap *bitmap;
    hipEvent_t start,stop;
    float totalTime;
    float frames;
}

//cptr is heater locations
__global__ void copy_const_kernel(float * iptr, const float *cptr)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * gridDim.x + blockDim.x;

    if(cptr[offset] != 0)
    {
        iptr[offset] = cptr[offset];
    }

}

__global__ void blend_kernel(float *outSrc, const float* inSrc)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * gridDim.x + blockDim.x;

    int left = offset - 1;
    int right = offset + 1;
    if(x == 0)
    {
        left++;
    }
    if(x == DIM -1)
    {
        right--;
    }

    int top = offset - DIM;
    int bottom = offset + DIM;

    if(y == 0)
    {
        top+= DIM;
    }
    if(y == DIM-1)
    {
        bottom -= DIM;
    }

    outSrc[offset] = inSrc[offset] + SPEED * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - inSrc[offset] * 4);

}

void anim_gpu( uchar4* outputBitmap, DataBlock *d, int ticks ) {
      hipEventRecord( d->start, 0 ) );
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);

    // since tex is global and bound, we have to use a flag to
    // select which is in/out per iteration
    volatile bool dstOut = true;
    for (int i=0; i<90; i++) {
        float   *in, *out;
        if (dstOut) {
            in  = d->dev_inSrc;
            out = d->dev_outSrc;
        } else {
            out = d->dev_inSrc;
            in  = d->dev_outSrc;
        }
        copy_const_kernel<<<blocks,threads>>>( in );
        blend_kernel<<<blocks,threads>>>( out, dstOut );
        dstOut = !dstOut;
    }

    float_to_color<<<blocks,threads>>>( outputBitmap,d->dev_inSrc );

    hipEventRecord( d->stop, 0 );
    hipEventSynchronize( d->stop );
    float   elapsedTime;
    hipEventElapsedTime( &elapsedTime,d->start, d->stop );
    d->totalTime += elapsedTime;
    ++d->frames;
    printf( "Average Time per frame:  %3.1f ms\n",d->totalTime/d->frames  );
}

// clean up memory allocated on the GPU
void anim_exit( DataBlock *d )
{
      hipUnbindTexture( texIn );
      hipUnbindTexture( texOut );
      hipUnbindTexture( texConstSrc );
      hipFree( d->dev_inSrc );
      hipFree( d->dev_outSrc );
      hipFree( d->dev_constSrc );

      hipEventDestroy( d->start );
      hipEventDestroy( d->stop );
}

int main(void)
{
    Data

}
