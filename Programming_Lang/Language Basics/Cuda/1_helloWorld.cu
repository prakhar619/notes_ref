//Host -> CPU AND RAM
//DEVICE -> GPU and its internal memory



#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void kernel( void )	//alerts the compiler that a function should run on a device instead of the host
{
}
//nvcc gives kernel() to compiler that handles device code
int main(void) 
{
	kernel<<<1,1>>>();	//gives parameter to compiler during runtime
	printf( "Hello, World!\n" );
	return 0;
}
