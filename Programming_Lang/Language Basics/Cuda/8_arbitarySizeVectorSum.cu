
#include <hip/hip_runtime.h>
#include <stdio.h>

#define  N 100000

__global__ void add(int* a,int *b,int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while(tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}
int main()
{
    int threadPerBlock = 1000;
    int blockCount = 10;


    int a[N], b[N], c[N];
    int* dev_a, *dev_b, *dev_c;

    hipMalloc(&dev_a,sizeof(int)*N);
    hipMalloc(&dev_b,sizeof(int)*N);
    hipMalloc(&dev_c,sizeof(int)*N);

    for(int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = 3*i -1;
    }

    hipMemcpy(dev_a,a,sizeof(int)*N,hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,sizeof(int)*N,hipMemcpyHostToDevice);

    add<<<blockCount,threadPerBlock>>>(dev_a,dev_b,dev_c);

    hipMemcpy(c,dev_c,sizeof(int)*N,hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++)
    {
        if(a[i] + b[i] != c[i])
            printf("ERROR");
        else
        printf("%d + %d = %d\n",a[i],b[i],c[i]);
    }
    return 1;
}
